// BLAS:Basic Linear Algebra Subprograms
#include <hipblas.h>
// exception用
#include <stdexcept>
#include <iostream>
#include "cuMat.h"
using namespace std;

/*
コピーコンストラクタです。既存のcuMatオブジェクトを引数に取り、そのオブジェクトと同じ内容の新しいcuMatオブジェクトを作成します。
コピーコンストラクタの特徴は:
  クラス自身の型の参照を1つだけ引数に取る
  引数はconst参照である
  新しいオブジェクトを引数のオブジェクトと同じ状態に初期化する
コピーコンストラクタは、以下のような場面で暗黙的に呼び出されます:
  関数の引数渡しでオブジェクトがコピーされるとき
  関数の戻り値でオブジェクトが返されるとき
  オブジェクトを別のオブジェクトで初期化するとき (例: cuMat mat2 = mat1;)
*/
cuMat(int rows, int cols)
{
    hipblasCreate(&cudaHandle);
    cudThreadSynchronize();
    new matrix(rows, cols);
}

cuMat::cuMat(const cuMat &a) : rows(a.rows), cols(a.cols)
{
    hipblasCreate(&cudaHandle);
    // ホストとデバイスの両方からアクセス可能なメモリを割り当て
    hipMallocManaged(&mDevice, rows * cols * sizeof(float));
    mHost = mDevice;
    hipDeviceSynchronize();

    hipError_t error = hipMemcpy(mDevice, a.mDevice, 
                rows * cols * sizeof(float),
                hipMemcpyDeviceToDevice);
    
    if (error != hipSuccess)
    {
        // エラー処理を行う
        throw std::exception("hipMemcpy failed!");
    }
}

cuMat::~cuMat()
{
    hipFree(mDevice);
    hipblasDestroy(cudaHandle);
}

void cuMat::new_matrix(int rows, int cols)
{
    
}

void cuMat::print() const
{
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            std::cout << mHost[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

void cuMat::plus(const cuMat &b, cuMat &r)
{
    float alpha = 1;
    float beta = 1;

    /*
    hipblasSgeam:
        C=α⋅A+β⋅B
        ここで、A と B は入力行列、C は結果行列、alpha と beta はスカラー値です。

    引数:
    r.cudaHandle: CUBLASライブラリのハンドル。
    HIPBLAS_OP_N: 行列の転置を行わないことを示します。
    rows: 行列の行数。
    cols: 行列の列数。
    &alpha: スカラー値 alpha のポインタ。
    mDevice: 元の行列のデータを指すデバイスポインタ。
    rows: 元の行列のリーディングディメンション（行数）。
    &beta: スカラー値 beta のポインタ。
    b.mDevice: 加算する行列 b のデータを指すデバイスポインタ。
    rows: 加算する行列 b のリーディングディメンション（行数）。
    r.mDevice: 結果を格納する行列 r のデータを指すデバイスポインタ。
    r.rows: 結果を格納する行列 r のリーディングディメンション（行数）。
    */
    hipblasStatus_t stat = hipblasSgeam(r.cudaHandle, HIPBLAS_OP_N,
        HIPBLAS_OP_N, rows, cols, &alpha, mDevice, rows, &beta,
        b.mDevice, rows, r.mDevice, r.rows);
    
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout << "cannot hipblasSgeam" << endl;
    }
    hipDeviceSynchronize();
}

cuMat& cuMat::operator=(const cuMat &a) {
    new_matrix(a.rows, a.cols);

    hipError_t error = hipMemcpy(mDevice, a.mDevice,
            rows * cols * sizeof(*mDevice), hipMemcpyDeviceToDevice);
    if (error != hipSuccess)
        printf("cuMat operator= hipMemcpy error\n");

    return *this;
}

cuMat operator+(const cuMat &a, const cuMat &b) {
    cuMat r = a;
    r.plus(b, r);

    return r;
}
