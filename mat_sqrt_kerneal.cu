#include "hip/hip_runtime.h"
#include "mat_sqrt_kernel.h"
#define BLOCK_SIZE 32

/*
device:
    デバイス(GPU)上で実行される関数につける修飾子
    __device__関数は、カーネル関数(__global__関数)や他の__device__関数から呼び出すことができる
    ホスト(CPU)コードからは直接呼び出せない
global:
    カーネル関数につける修飾子。カーネル関数はデバイス(GPU)上で実行される
    ホスト(CPU)コードから呼び出すことができる。その際、実行設定(グリッド、ブロックサイズなど)を指定する
    __global__関数からは、他の__global__関数を呼び出せない
    戻り値の型はvoidでなければならない
*/

// GPU上で実行,インライン化を強制
__device__ __forceinline__ float mat_sqrt (float a, float alpha){
    return std::sqrt(a+alpha);
}

// カーネル関数を定義
/*
__restrict__ を使用することで、プログラマはコンパイラに対して「このポインタが指すメモリ領域は他のポインタと重ならない」と保証します。
これにより、コンパイラはより積極的な最適化を行うことができます。
*/
__global__ void mat_sqrt_kernel(const float* __restrict__ src, float* __restrict__ dst, int m, int n, float alpha) {
    // Block Index:グリッド内のスレッドブロックのインデックスを示します。
    // Block Dimension:1つのスレッドブロック内のスレッドの数
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        dst[row * n + col] = mat_sqrt(src[row * n + col], alpha);
    }
}

void mat_sqrt_kernel_exec(const float *src, float *dst, int m, int n, float alpha){
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    // カーネル関数の完了を待機
    mat_sqrt_kernel<<<grid, block>>>(src, dst, m, n, alpha);
    hipDeviceSynchronize();
}
